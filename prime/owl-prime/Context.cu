#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
// SPDX-License-Identifier: Apache-2.0



#include "owl-prime/Context.h"
#include "owl-prime/Triangles.h"
#include "owl-prime/Spheres.h"
#include "owl-prime/Group.h"

primer::Context *primer::Context::createOffloadContext(int gpuID)
{ return new op::Context(gpuID); }

extern "C" char deviceCode_ptx[];

namespace op {
  
  OWLVarDecl Context::lpVariables[]
  = {
     { "rays",      OWL_RAW_POINTER, OWL_OFFSETOF(Context::LPData,rays) },
     { "hits",      OWL_RAW_POINTER, OWL_OFFSETOF(Context::LPData,hits) },
     { "activeIDs", OWL_RAW_POINTER, OWL_OFFSETOF(Context::LPData,activeIDs) },
     { "model",     OWL_GROUP,       OWL_OFFSETOF(Context::LPData,model) },
     { "numRays",   OWL_INT,         OWL_OFFSETOF(Context::LPData,numRays) },
     { "flags",     OWL_ULONG,       OWL_OFFSETOF(Context::LPData,flags) },
     { nullptr /* end of list sentinel */ }
  };

  Context::Context(int gpuID)
  {
    if (gpuID < 0) gpuID = 0;
    
    owl = owlContextCreate(&gpuID,1);
    module = owlModuleCreate(owl,deviceCode_ptx);
    rayGen = owlRayGenCreate(owl,module,"traceRays",sizeof(int),nullptr,0);
    launchParams = owlParamsCreate(owl,sizeof(LPData),lpVariables,-1);
    
    meshGeomType = owlGeomTypeCreate(owl,OWL_TRIANGLES,
                                     sizeof(Triangles::SBTData),
                                     Triangles::variables,-1);
    owlGeomTypeSetClosestHit(meshGeomType,0,module,"TrianglesCH");
    owlGeomTypeSetAnyHit(meshGeomType,0,module,"TrianglesAH");

#if 0
    // disabled right now, not hooked up to API, yet
    spheresGeomType = owlGeomTypeCreate(owl,OWL_GEOM_USER,
                                        sizeof(Spheres::SBTData),
                                        Spheres::variables,-1);
    owlGeomTypeSetBoundsProg(spheresGeomType,module,"Spheres");
    owlGeomTypeSetIntersectProg(spheresGeomType,0,module,"Spheres");
    owlGeomTypeSetClosestHit(spheresGeomType,0,module,"Spheres");
#endif
    owlBuildPrograms(owl);
    owlBuildPipeline(owl);
  }
  
  void Context::checkSBT()
  {
    if (!sbtDirty) return;

    owlBuildSBT(owl);
    owlBuildPipeline(owl);
    sbtDirty = false;
  }

  template<typename T>
  inline __both__
  const T &getWithOffset(const T *base, int idx, size_t strideInBytes)
  {
    unsigned char *ptr = (unsigned char *)base;
    ptr += idx * strideInBytes;
    return *(T*)ptr;
  }

  __global__ void copySpheres(float4 *spheres,
                              int numSpheres,
                              const float *x,
                              const float *y,
                              const float *z,
                              int centerStride,
                              const float *r,
                              int rStride)
  {
    int tid = threadIdx.x+blockIdx.x*blockDim.x;
    if (tid >= numSpheres) return;
    spheres[tid]
      = {
         getWithOffset(x,tid,centerStride),
         getWithOffset(y,tid,centerStride),
         getWithOffset(z,tid,centerStride),
         getWithOffset(r,tid,rStride)
    };
  }

  /*! create a mesh from vertex array and index array */
  primer::Geom *Context::createTriangles(uint64_t userID,
                                         /* vertex array */
                                         const vec3f *vertices,
                                         size_t numVertices,
                                         size_t vertexStrideInBytes,
                                         /* index array */
                                         const vec3i *indices,
                                         size_t numIndices,
                                         size_t indexStrideInBytes)
  {
    // TODO: do all this without copies if these are already device pointers
    OWLBuffer vertexBuffer
      = owlManagedMemoryBufferCreate(owl,OWL_FLOAT3,numVertices,0);
    vec3f *d_vertices = (vec3f*)owlBufferGetPointer(vertexBuffer,0);
    
    OWLBuffer indexBuffer
      = owlManagedMemoryBufferCreate(owl,OWL_INT3,numIndices,0);
    vec3i *d_indices = (vec3i*)owlBufferGetPointer(indexBuffer,0);

    for (int i=0;i<numVertices;i++) 
      d_vertices[i] = getWithOffset(vertices,i,vertexStrideInBytes);
    for (int i=0;i<numIndices;i++) 
      d_indices[i] = getWithOffset(indices,i,indexStrideInBytes);
    
    assert(size_t(int(numIndices)) == numIndices);
    assert(size_t(int(numVertices)) == numVertices);
    return new Triangles(this,userID,
                         vertexBuffer,(int)numVertices,
                         indexBuffer,(int)numIndices);
  }

  primer::Group *Context::createGroup(std::vector<OPGeom> &geoms) 
  {
    return new op::Group(this,geoms);
  }

  primer::Model *Context::createModel(const std::vector<OPGroup>  &groups,
                                      const std::vector<affine3f> &xfms)
  {
    return new op::Model(this,groups,xfms);
  }
  
} // ::op
