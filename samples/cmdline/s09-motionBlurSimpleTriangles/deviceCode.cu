#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
// SPDX-License-Identifier: Apache-2.0



#include "deviceCode.h"
#include <owl/common/math/random.h>
#include <optix_device.h>

typedef owl::common::LCG<4> Random;

OPTIX_RAYGEN_PROGRAM(simpleRayGen)()
{
  const RayGenData &self = owl::getProgramData<RayGenData>();
  const vec2i pixelID = owl::getLaunchIndex();
  if (pixelID == owl::vec2i(0)) {
    printf("%sHello OptiX From your First RayGen Program%s\n",
           OWL_TERMINAL_CYAN,
           OWL_TERMINAL_DEFAULT);
  }

  Random rng(pixelID);
  
  const vec2f screen = (vec2f(pixelID)+vec2f(.5f)) / vec2f(self.fbSize);
  owl::Ray ray;
  ray.origin    
    = self.camera.pos;
  ray.direction 
    = normalize(self.camera.dir_00
                + screen.u * self.camera.dir_du
                + screen.v * self.camera.dir_dv);
  
  vec3f avgColor = 0.f;
  const int numSPP = 16;
  for (int i=0;i<numSPP;i++) {
    ray.time = rng();
    vec3f color;
    owl::traceRay(/*accel to trace against*/self.world,
                  /*the ray to trace*/ray,
                  /*prd*/color);
    
    avgColor += color;
  }
  avgColor *= 1.f/numSPP;
  const int fbOfs = pixelID.x+self.fbSize.x*pixelID.y;
    self.fbPtr[fbOfs]
      = owl::make_rgba(avgColor);
  }
  
inline __device__ vec3f lerp(const vec3f &A, const vec3f &B, float t)
{ return (1.f-t)*A + t*B; }
  
OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)()
{
  vec3f &prd = owl::getPRD<vec3f>();

  const TrianglesGeomData &self = owl::getProgramData<TrianglesGeomData>();
  
  // compute normal:
  const int   primID  = optixGetPrimitiveIndex();
  const float time    = optixGetRayTime();
  const vec3i index   = self.index[primID];
  const vec3f &A0     = self.vertex0[index.x];
  const vec3f &B0     = self.vertex0[index.y];
  const vec3f &C0     = self.vertex0[index.z];
  const vec3f &A1     = self.vertex1[index.x];
  const vec3f &B1     = self.vertex1[index.y];
  const vec3f &C1     = self.vertex1[index.z];
  const vec3f A       = lerp(A0,A1,time);
  const vec3f B       = lerp(B0,B1,time);
  const vec3f C       = lerp(C0,C1,time);
  const vec3f Ng      = normalize(cross(B-A,C-A));

  const vec3f rayDir  = optixGetWorldRayDirection();
  prd = (.2f + .8f*fabs(dot(rayDir,Ng)))*self.color;
}

OPTIX_MISS_PROGRAM(miss)()
{
  const vec2i pixelID = owl::getLaunchIndex();

  const MissProgData &self = owl::getProgramData<MissProgData>();
  
  vec3f &prd = owl::getPRD<vec3f>();
  int pattern = (pixelID.x / 8) ^ (pixelID.y/8);
  prd = (pattern&1) ? self.color1 : self.color0;
}

